#include "hip/hip_runtime.h"

/*
* Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
*
* NVIDIA Corporation and its licensors retain all intellectual property and proprietary
* rights in and to this software, related documentation and any modifications thereto.
* Any use, reproduction, disclosure or distribution of this software and related
* documentation without an express license agreement from NVIDIA Corporation is strictly
* prohibited.
*
* TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
* AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
* INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
* PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
* SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
* LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
* BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
* INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
* SUCH DAMAGES
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

rtTextureSampler<float4, 2>  request_texture;

rtBuffer<float4, 2>          result_buffer;
rtBuffer<float4, 2>          position_buffer;

rtDeclareVariable(uint, shadow_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float3, light_pos, , );
rtDeclareVariable(rtObject, reflectors, , );
rtDeclareVariable(float3, eye_pos, , );
rtDeclareVariable(float3, eoc_eye_right_pos, , );
rtDeclareVariable(float3, eoc_eye_top_pos, , );
rtDeclareVariable(float3, rightND, , );
rtDeclareVariable(float3, topND, , );
rtDeclareVariable(optix::Matrix4x4, optixModeView_Inv, , );
rtDeclareVariable(float2, resolution, , );

rtDeclareVariable(optix::Matrix4x4, rightModelView, , );

rtDeclareVariable(optix::Matrix4x4, optixModelView, , );
struct PerRayData_shadow
{
	float3 attenuation;
	float3 worldPos;
	float t_hit;
};


rtDeclareVariable(float2, bbmin, , );
rtDeclareVariable(float2, bbmax, , );
__device__ float3 getImagePos(float2 tc)
{
	float2 xy = bbmin + (bbmax - bbmin)*tc;
	xy = xy;
	float4 temp = make_float4(xy.x, xy.y, -1, 1)*optixModeView_Inv;
	temp = temp / temp.w;
	return make_float3(temp.x, temp.y, temp.z);
}


RT_PROGRAM void shadow_request()
{
	float2 tc = make_float2(launch_index.x, launch_index.y) / resolution;
	float4 textValue = tex2D(request_texture, launch_index.x + 0.5, launch_index.y + 0.5);// texture x ͨ���洢����ǰ�����dis��Ϣ
	/*if (launch_index.y == 838)
	{
		result_buffer[launch_index] = make_float4(1,1,0, 1);
		return;

	}
	result_buffer[launch_index] = textValue;
	return;*/
	if (textValue.x >= 1.0)
	{
		float3 targetPos = make_float3(textValue.y, textValue.z, textValue.w);
		float3 ray_origin = eoc_eye_right_pos;
		PerRayData_shadow prd;
		prd.attenuation = make_float3(-1);
		float3 L = targetPos - ray_origin;
		float dist = sqrtf(dot(L, L));
		float3 ray_direction = L / dist;
		optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, shadow_ray_type, dist, textValue.x);
		rtTrace(reflectors, ray, prd);
		result_buffer[launch_index] = make_float4(prd.attenuation,1);
		result_buffer[launch_index].z =( result_buffer[launch_index].z +3 )/4;
		float3 worldPos = prd.worldPos;
		float3 rightCameraToWorldPos = worldPos - eoc_eye_right_pos;
		float dis = dot(rightCameraToWorldPos, rightND);
		float4 temp = make_float4(worldPos, 1)*rightModelView;
		result_buffer[launch_index].w = -temp.z;
		position_buffer[launch_index] = make_float4(worldPos,1);
		return;
	}
	else  	if (textValue.x <= -1.0)
	{
		float3 targetPos = make_float3(textValue.y, textValue.z, textValue.w);
		float3 ray_origin = eoc_eye_top_pos;
		PerRayData_shadow prd;
		prd.attenuation = make_float3(-1);
		float3 L = targetPos - ray_origin;
		float dist = sqrtf(dot(L, L));
		float3 ray_direction = L / dist;
		optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, shadow_ray_type, dist, -textValue.x);
		rtTrace(reflectors, ray, prd);
		result_buffer[launch_index] = make_float4(prd.attenuation, 1);
		result_buffer[launch_index].y = (result_buffer[launch_index].y + 3) / 4;
		float3 worldPos = ray_origin + ray_direction*prd.t_hit;
		float3 topCameraToWorldPos = worldPos - eoc_eye_top_pos;
		float dis = dot(topCameraToWorldPos, topND);
		result_buffer[launch_index].w = -dis;
		position_buffer[launch_index] = make_float4(worldPos, 1);
		return;
	}
	result_buffer[launch_index] = textValue;
	position_buffer[launch_index] = make_float4(0,0,0, 1);
}

RT_PROGRAM void exception()
{
}
