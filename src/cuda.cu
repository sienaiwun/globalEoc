#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>
#include <nvMatrix.h>
#include "Camera.h"

#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif



#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define INTERSECT 1
#define OHTEROBJECT (-1)
#define MISSINGNOTE 0
#define NOOBJECTNOTE 2
#define OUTOCCLUDED 3
#define FLT_MAX 99999.9
//�󽻵ĺ�
#define RAYISUP 0
#define RAYOUT 1
#define RAYISUNDER 2
texture<float4, 2, hipReadModeElementType> cudaEdgeTex;//��¼����Edge,edge����x��¼x��sobal,edge����y��¼y��sobal
texture<float4, 2, hipReadModeElementType> cudaOccuderTex;
texture<float4, 2, hipReadModeElementType> cudaTopOccuderTex;
texture<float4, 2, hipReadModeElementType> cudaColorTex;
texture<float4, 2, hipReadModeElementType> cudaPosTex;
texture<float4, 2, hipReadModeElementType> cudaNormalTex;
texture<float4, 2, hipReadModeElementType> optixColorTex;
texture<float4, 2, hipReadModeElementType> posBlendTex;
hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
typedef enum {
	isVolumn,
	notVolumn,
}pixelEnum;



uint3 *cuda_PBO_Buffer;

__device__ uint3* d_cudaPboBuffer;
float4 *cuda_TexturePbo_buffer, *cuda_top_TexturePbo_buffer;
__device__ float4* d_cudaTexture;
__device__ float4* d_cudaTopTexture;
__device__ int d_imageWidth, d_imageHeight, d_outTextureWidth, d_outTextureHeigh, d_outTopTextureWidth, d_outTopTextureHeight, d_construct_width, d_construct_height;
__device__ int d_index;
__device__ ListNote* d_listBuffer;
__device__ ListNote* d_listBuffer_top;
__device__ int d_atomic;
__device__ float3 d_cameraPos;
__device__ float3 d_eocPos;
__device__ float3 d_eocTopPos;
__device__ float3 d_rightRD;
__device__ float3 d_topRD;
float* modelView;
__device__ float* d_modelView;
float* proj;
__device__ float* d_porj;


float* modelViewRight;
__device__ float* d_modelViewRight;

float* modelView_construct;
float* project_construct;
float* modelView_inv;
__device__ float* d_modelView_construct;
__device__ float* d_project_construct;
__device__ float3 d_construct_cam_pos;
__device__ float* d_modeView_inv_construct;
__device__ float2 d_bbmin, d_bbmax;


__device__ float4* d_cuda_construct_texture;
float4 *cuda_construct_texturePbo_buffer;

__device__ float4* d_map_buffer;  //d_map_buffer x ��¼����texture ���µ�texture��ӳ�䣬y��¼�����ڵ����صĵ������������ĵ�����ӳ�䣬z��¼�����ڵ�������noteId
float4* cuda_map_buffer;
__device__ int nearestInt(float value)
{
	return value + 0.5;
}
__device__ int startInt(float value, bool isUp)
{
	if (isUp)
		return floor(value);
	else
		return ceil(value);
}
__host__ __device__ float4 MutiMatrix(float * Matrix, float4 invalue)
{
	float x = invalue.x;
	float y = invalue.y;
	float z = invalue.z;
	float w = invalue.w;

	float outx = x*Matrix[0] + y*Matrix[4] + z*Matrix[8] + w*Matrix[12];
	float outy = x*Matrix[1] + y*Matrix[5] + z*Matrix[9] + w*Matrix[13];
	float outz = x*Matrix[2] + y*Matrix[6] + z*Matrix[10] + w*Matrix[14];
	float outw = x*Matrix[3] + y*Matrix[7] + z*Matrix[11] + w*Matrix[15];

	return make_float4(outx, outy, outz, outw);
}
__host__ __device__ float  element(float* _array, int row, int col)
{
	return _array[row | (col << 2)];
}
__host__ __device__ float4 MutiMatrixN(float * Matrix, float4 invalue)
{
	float x = invalue.x;
	float y = invalue.y;
	float z = invalue.z;
	float w = invalue.w;
	float r[4];
	for (int i = 0; i < 4; i++)
	{
		r[i] = (x * element(Matrix, i, 0) + y * element(Matrix, i, 1) + z * element(Matrix, i, 2) + w * element(Matrix, i, 3));
	}
	return make_float4(r[0], r[1], r[2], r[3]);
}
__host__ __device__ void MutiMatrix(float* src, float* matrix, float* r)
{
	for (int i = 0; i < 4; i++)
	{
		r[i] = (src[0] * element(matrix, i, 0) + src[1] * element(matrix, i, 1) + src[2] * element(matrix, i, 2) + src[3] * element(matrix, i, 3));
	}

}
__device__ float getRatioInSpan(float3 beginPos, float3 endPos, float* p_modelView, float3 testPos);
__device__ float3 projective_interpo(float3 beginPos, float3 endPos, float* p_modelView, float ratio, int print);
/*
��¼���ߺ��������󽻵Ľ�����󽻵Ľ����zֵ������¼�� proj_ratio ��
*/
__device__ bool rayIntersertectTriangle(float3 origin, float3 directionN, float3 cameraPos, float3 edgePoint1/*beginPos*/, float3 edgePoint2/*endPos*/, float* modelView_float, float noteSpan,float3* pIntersectWorld3, float3* pLineIntersect, bool& isOnrTiangle, float& proj_ratio, float3& reversePoint3)
{
	//printf("origin:(%f,%f,%f)\n", origin.x, origin.y, origin.z);
	//printf("directionN:(%f,%f,%f)\n", directionN.x, directionN.y, directionN.z);

	const float3 e0 = edgePoint1 - cameraPos;
	const float3 e1 = cameraPos - edgePoint2;
	const float3 e2 = edgePoint2 - edgePoint1;
	const float3 n = normalize(cross(e1, e0));
	float3 toIntersection, ratio3, lineIntersect;
	if (1)
	{
		toIntersection = directionN * dot(cameraPos - origin, n) / dot(n, directionN);
		const float3 intersectPos = origin + toIntersection;
		const float3 lineNormal2 = normalize(cross(normalize(e2), n));
		const float3 tolineIntersectPoint = (intersectPos - cameraPos)* dot(edgePoint1 - cameraPos, lineNormal2) / dot(intersectPos - cameraPos, lineNormal2);
		lineIntersect = cameraPos + tolineIntersectPoint;
		isOnrTiangle = false;
		*pIntersectWorld3 = intersectPos;
	}
	else
	{
		const float3 lineNormal2 = normalize(cross(normalize(e2), n));
		const float3 tolineIntersectPoint = directionN* dot(edgePoint1 - origin, lineNormal2) / dot(directionN, lineNormal2);
		//printf("tolineIntersectPoint:(%f,%f,%f)\n", tolineIntersectPoint.x, tolineIntersectPoint.y, tolineIntersectPoint.z);
		lineIntersect = origin + tolineIntersectPoint;
		isOnrTiangle = true;
		*pIntersectWorld3 = lineIntersect;

	}
	//printf("intersect:(%f,%f,%f)\n", lineIntersect.x, lineIntersect.y, lineIntersect.z);

	*pLineIntersect = lineIntersect;
	ratio3 = (lineIntersect - edgePoint1) / (edgePoint2 - edgePoint1);
	//printf("edge1:%f,%f,%f, edge2:%f,%f,%f\n", edgePoint1.x, edgePoint1.y, edgePoint1.z, edgePoint2.x, edgePoint2.y, edgePoint2.z);
	proj_ratio = getRatioInSpan(edgePoint1, edgePoint2, modelView_float, lineIntersect);
	//printf("linar(%f,%f,%f),rePorj_value:%f\n", ratio3.x, ratio3.y, ratio3.z, proj_ratio);

	//printf("proj_ratio:%f\n", proj_ratio);
	reversePoint3 = projective_interpo(edgePoint1, edgePoint2, d_modelViewRight, proj_ratio, 1);
	//printf("ratio3:%f,%f,%f\n", ratio3.x, ratio3.y, ratio3.z);
	//printf("dotValue:%f\n", dot(n, directionN));
	float lgap = 0.5 / noteSpan;
	//printf("noteSpane:%f\n", noteSpan);
	float lmin = 0 - lgap;
	float lmax = 1 + lgap;
	//printf("lmin,lmax:%f,%f\n", lmin, lmax);
	if (lmin < ratio3.x && ratio3.x <= lmax && lmin < ratio3.y && ratio3.y <= lmax && lmin < ratio3.z && ratio3.z <= lmax)
	{
		//printf("touched\n");
		return TRUE;
	}

	//printf("no touched\n");
	return FALSE;

}
__device__ int2 nearestTc(float2 tc)
{
	return make_int2(tc.x, tc.y);//ֱ�ӽ���intת������Ϊ��ȥ0.5+0.5
}
//ӳ�䵽Ť���ռ�
__device__ float4 colorTextreNorTc(float2 tc)
{
	float2 nonNorTc = tc* make_float2(d_imageWidth, d_imageHeight);
	int2 mapTx = nearestTc(nonNorTc);
	int index = mapTx.y * d_imageWidth + mapTx.x;
	int mappedX = (int)(d_map_buffer[index].x + 0.5);

	//printf("mapped tc:(%d,%f),z:(%f)\n", mappedX, nonNorTc.y, tex2D(optixColorTex, nonNorTc.x, nonNorTc.y).z);
	return tex2D(optixColorTex, mappedX, nonNorTc.y);
}
__device__ int getNoteIndex(float2 tc)
{
	//printf("in note id\n");
	float2 nonNorTc = tc* make_float2(d_imageWidth, d_imageHeight);

	int2 mapTx = nearestTc(nonNorTc);
	//printf("mapped tc:(%d,%d)\n", mapTx.x, mapTx.y);
	int index = mapTx.y * d_imageWidth + mapTx.x;
	int noteId = (int)(d_map_buffer[index].z + 0.5);
	//printf("noteId tc:(%d)\n", noteId);
	return noteId;
}

__device__ bool isOccluedeArea(float2 tc)
{
	float2 nonNorTc = tc* make_float2(d_imageWidth, d_imageHeight);
	int2 mapTx = nearestTc(nonNorTc);
	int index = mapTx.y * d_imageWidth + mapTx.x;
	int mappedY = (int)(d_map_buffer[index].y + 0.5);
	if (mappedY < 1)
	{
		//printf("not occluded\n");
		return false;
	}
	//printf("occluded mapped coord:(%d,%d)\n", mappedY, mapTx.y);
	return true;

}
__device__ bool canGetMappedPosition(float2 tc, float4* poc)
{
	float2 nonNorTc = tc* make_float2(d_imageWidth, d_imageHeight);
	int2 mapTx = nearestTc(nonNorTc);
	int index = mapTx.y * d_imageWidth + mapTx.x;
	int mappedX = (int)(d_map_buffer[index].y + 0.5);
	if (mappedX < 1)
	{
		return false;
	}
	//printf("mapped coord:(%d,%d)\n", mappedY, mapTx.y);
	*poc = tex2D(optixColorTex, mappedX, nonNorTc.y);
	return true;

}
__device__ bool noMappedPosition(float2 tc, float4* poc)
{
	float2 nonNorTc = tc;
	int2 mapTx = nearestTc(nonNorTc);
	int index = mapTx.y * d_imageWidth + mapTx.x;
	int mappedX = (int)(d_map_buffer[index].y + 0.5);

	//printf("mapped coord:(%d,%d)\n", mappedX, mapTx.y);
	*poc = tex2D(optixColorTex, mappedX, nonNorTc.y);
	//printf("color:(%f,%f,%f,%f)\n", poc->x, poc->y, poc->z, poc->w);
	if (poc->w < 0)
	{
		return false;
	}
	if (mappedX < 1)
	{
		return false;
	}
	//printf("%f,%f,%f,%f\n", poc->x, poc->y, poc->z, poc->w);

	return true;

}
__host__ __device__ void MutiMatrix(float * Matrix, float x, float y, float z, float &outx, float &outy, float &outz)
{
	float tempx = x*Matrix[0] + y*Matrix[1] + z*Matrix[2] + Matrix[3];
	float tempy = x*Matrix[4] + y*Matrix[5] + z*Matrix[6] + Matrix[7];
	float tempz = x*Matrix[8] + y*Matrix[9] + z*Matrix[10] + Matrix[11];
	float tempt = x*Matrix[12] + y*Matrix[13] + z*Matrix[14] + Matrix[15];
	float3 result;
	if (tempt<0.0001)
	{
		outx = FLT_MAX;
		outy = FLT_MAX;
		outz = FLT_MAX;
	}
	outx = tempx / tempt;
	outy = tempy / tempt;
	outz = tempz / tempt;
}
__device__ float2 getCameraTc(float3 pos,float* modelMat,float* projMat)
{
	float4 temp = MutiMatrixN(modelMat, make_float4(pos, 1));
	temp = MutiMatrixN(projMat, temp);
	temp = temp / temp.w;
	float2 tc;
	tc.x = 0.5*temp.x + 0.5;
	tc.y = 0.5*temp.y + 0.5;
	return tc;
}

class List
{

};
__device__ float repo(float value)
{
	return 1.0f / value;
}
__device__ float getRatioInSpan(float3 beginPos, float3 endPos, float* p_modelView, float3 testPos)
{
	float x, y, z1, z2, z3;
	float4 temp = MutiMatrixN(p_modelView, make_float4(beginPos.x, beginPos.y, beginPos.z, 1));
	z1 = temp.z;
	temp = MutiMatrixN(p_modelView, make_float4(endPos.x, endPos.y, endPos.z, 1));
	z2 = temp.z;
	temp = MutiMatrixN(p_modelView, make_float4(testPos.x, testPos.y, testPos.z, 1));
	z3 = temp.z;
	float real_ratio = (repo(z1) - repo(z3)) / (repo(z1) - repo(z2));
	/*
	printf("in projection\n");
	printf("beginPos:(%f,%f,%f)\n", beginPos.x, beginPos.y, beginPos.z);
	printf("endPos:(%f,%f,%f)\n", endPos.x, endPos.y, endPos.z);
	printf("testPos:(%f,%f,%f)\n", testPos.x, testPos.y, testPos.z);
	printf("z1:%f,z2:%f,z3:%f\n",z1,z2,z3);
	printf("repo z1:%f,z2:%f,z3:%f\n", repo(z1), repo(z2), repo(z3));
	*/
	//printf("ration:%f\n", real_ratio);

	return real_ratio;
	return (z3 - z2) / (z1 - z2);
}
__device__ float3 projective_interpo(float3 beginPos, float3 endPos, float* p_modelView, float ratio, int print = 0)
{
	float x, y, z1, z2;
	float4 temp = MutiMatrixN(p_modelView, make_float4(beginPos, 1));
	z1 = temp.z;
	temp = MutiMatrixN(p_modelView, make_float4(endPos, 1));
	z2 = temp.z;
	float real_z = repo((1 - ratio) *repo(z1) + ratio* repo(z2));
	float real_ratio = (real_z - z1) / (z2 - z1);
	if (print)
	{
		//	 printf("z1:%f,z2:%f,ratio:%f,real_z:%f,real_ratio:%f\n",z1,z2,ratio, real_z,real_ratio);

	}
	return beginPos*(1 - real_ratio) + endPos  *  real_ratio;
}
__device__ bool isVolume(float2 uv)
{
	float4 value = tex2D(cudaOccuderTex, uv.x, uv.y);
	return value.x > 0.5;
}
__device__ bool isVolumeTop(float2 uv)
{
	float4 value = tex2D(cudaTopOccuderTex, uv.x, uv.y);
	return value.x > 0.5;

}
__device__ bool isEdge(float2 uv)
{
	return tex2D(cudaEdgeTex, uv.x, uv.y).x > 0.05;
}
__device__ bool isEdgeTop(float2 uv)
{
	return tex2D(cudaEdgeTex, uv.x, uv.y).y > 0.05;
}
__device__ bool isMinusEdge(float2 uv)
{
	return tex2D(cudaEdgeTex, uv.x, uv.y).x < -0.05;
}
__device__ bool isMinusEdgeTop(float2 uv)
{
	return tex2D(cudaEdgeTex, uv.x, uv.y).y < -0.05;
}
__device__ bool isTracingEdge(float2 tc)
{
	float2 nonNorTc = tc* make_float2(d_imageWidth, d_imageHeight);
	return isEdge(nonNorTc) || isEdgeTop(nonNorTc) || isMinusEdge(nonNorTc) || isMinusEdgeTop(nonNorTc);
}

__device__ float2 toUv(int x, int y)
{
	return make_float2(x + 0.5, y + 0.5);
}
__global__ void countRowKernelTop(int kernelWidth, int kernelHeight)
{
	int index = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	if (index > kernelWidth)
		return;
	//if (index != 512)
	//	return;
	int arrayNum = index;
	int accumNum = 0;
	int state = 0;
	pixelEnum etype = notVolumn;
	unsigned int* nextPtr = &d_cudaPboBuffer[arrayNum].x;
	int listIndex;
	int lastMinusIndey = 0;
	for (int y = 0; y< d_imageHeight; y++)
	{
		float2 currentUv = toUv(index, y);
		if (isMinusEdgeTop(currentUv))
		{
			lastMinusIndey = y;
		}
		if (isVolumeTop(currentUv) && etype == notVolumn)
		{
			//printf("insert :%d\n", y);
			listIndex = atomicAdd(&d_atomic, 1);
			atomicExch(nextPtr, listIndex);// write listIndex to next slot
			d_listBuffer[listIndex].beginIndex = y;
			d_listBuffer[listIndex].endIndex = y;
			d_listBuffer[listIndex].nextPt = 0;
			d_listBuffer[listIndex].leftEdge = lastMinusIndey;
			nextPtr = (unsigned int *)(&(d_listBuffer[listIndex].nextPt));

			etype = isVolumn;
		}
		else if (isVolumeTop(currentUv) && etype == isVolumn)
		{


		}
		else if (etype == isVolumn && isEdgeTop(currentUv))
		{
			d_listBuffer[listIndex].endIndex = y - 1;
			etype = notVolumn;
		}
	}
}
// ��¼interval  interval��leftEdge ��¼����ߵ�edge��Ϊ��������Ⱦ�ã���beginIndex��¼��һ���ڵ������ء�endIndex ��¼�����ұ߽磨�ұ߽߱�����-1λ��
// ÿһ��
__global__ void countRowKernel(int kernelWidth, int kernelHeight)
{
	int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	if (y > kernelHeight)
		return;
	//if (y != 838)
	//	return;
	int arrayNum = y;
	int accumNum = 0;
	int state = 0;
	pixelEnum etype = notVolumn;
	unsigned int* nextPtr = &d_cudaPboBuffer[arrayNum].x;
	int listIndex;
	int lastMinusIndex = 0;
	for (int x = 0; x < d_imageWidth; x++)
	{

		float2 currentUv = toUv(x, y);
		//if (x == 340)

		if (isMinusEdge(currentUv))
		{
			lastMinusIndex = x;
		}
		if (isVolume(currentUv) && etype == notVolumn)
		{
			//printf("insert :%d\n", x);
			listIndex = atomicAdd(&d_atomic, 1);
			atomicExch(nextPtr, listIndex);// write listIndex to next slot
			d_listBuffer[listIndex].beginIndex = x;
			d_listBuffer[listIndex].endIndex = x;
			d_listBuffer[listIndex].nextPt = 0;
			d_listBuffer[listIndex].leftEdge = lastMinusIndex;
			nextPtr = (unsigned int *)(&(d_listBuffer[listIndex].nextPt));
			etype = isVolumn;

		}
		else if (isVolume(currentUv) && etype == isVolumn)
		{


		}
		else if (etype == isVolumn && isEdge(currentUv))
		{
			//printf("end :%d\n", x);

			d_listBuffer[listIndex].endIndex = x - 1;
			//d_listBuffer[listIndex].endIndex = d_listBuffer[listIndex].beginIndex + 117;
			etype = notVolumn;
		}

	}
}
__device__ float  myfmax(float a, float b) {
	return ((a) > (b) ? a : b);
}
__device__ float distance(float3 leftPos, float3 currentPos, float3 eocCamera)
{
	float3 line1 = normalize(currentPos - eocCamera);

	float3 line2 = normalize(leftPos - d_cameraPos);
	float3 zhijiao = normalize(cross(line1, line2));
	float3 cuixian = normalize(cross(line2, zhijiao));
	/*printf("currentPos:(%f,%f,%f)\n", currentPos.x, currentPos.y, currentPos.z);
	printf("line1:(%f,%f,%f)\n", line1.x, line1.y, line1.z);
	printf("line2:(%f,%f,%f)\n", line2.x, line2.y, line2.z);
	printf("zhijiao:(%f,%f,%f)\n", zhijiao.x, zhijiao.y, zhijiao.z);
	printf("cuixian:(%f,%f,%f)\n", cuixian.x, cuixian.y, cuixian.z);
	printf("dot:(%f)\n", dot(line1, cuixian));*/
	float dis = (dot(d_cameraPos - eocCamera, cuixian) / (dot(line1, cuixian)));
	if (dis < 0)
		return 1000.0;
	else
		return myfmax(dis, 1.0);

}
__device__ void FillLine(int x)
{
	for (int y = 0; y < d_outTextureHeigh; y++)
	{
		int index = y*d_outTopTextureWidth + x;
		d_cudaTopTexture[index] = d_cudaTexture[y*d_outTextureWidth + x];
	}
}
__device__ float4 FillPoint(int x, int y)
{
	int index = y*d_outTextureWidth + x;
	return  d_cudaTexture[index];
}
__device__ void FillVolumnTop(int beginY, int endY, int x, int endUv, int leftEdge)
{
	int top = min(endY, d_outTopTextureHeight);
	//printf("volumn begin:%d,end:%d,top:%d\n",beginX,endX,top);
	float3 beforePos = make_float3(tex2D(cudaPosTex, x, endUv - 0.5));

	float3 endPos = make_float3(tex2D(cudaPosTex, x, endUv + 0.5));
	float3 leftEdgePos = make_float3(tex2D(cudaPosTex, x, leftEdge + 1.5));
	//printf("endPos:(%f,%f,%f)\n", endPos.x, endPos.y, endPos.z);
	///printf("beforePos:(%f,%f,%f)\n", beforePos.x, beforePos.y, beforePos.z);
	//printf("leftEdgePos:(%f,%f,%f)\n", leftEdgePos.x, leftEdgePos.y, leftEdgePos.z);
	//printf("camera:(%f,%f,%f)\n", d_cameraPos.x, d_cameraPos.y, d_cameraPos.z);
	float3 ecoCamera = d_eocTopPos;
	//printf("eoc:(%f,%f,%f)\n", ecoCamera.x, ecoCamera.y, ecoCamera.z);

	//for (int i = 0; i < 4; i++)
	//	printf("(%f,%f,%f,%f)\n", d_modelView[4 * i + 0], d_modelView[4 * i + 1], d_modelView[4 * i + 2], d_modelView[4 * i + 3]);
	for (int y = beginY; y < top; y++)
	{
		float ratio = (y * 1.0f - beginY*1.0f) / (top - 1 - beginY);
		float3 realPos = projective_interpo(beforePos, endPos, d_modelView, ratio);
		int index = y*d_outTopTextureWidth + x;
		float dis = distance(leftEdgePos, realPos, ecoCamera);
		d_cudaTopTexture[index] = make_float4(-dis, realPos.x, realPos.y, realPos.z);
	}
}
__device__ void FillSpanTop(int beginY, int endY, int x, float2 beginUv, float2 endUv)
{
	int top = min(endY, d_outTopTextureHeight);
	//printf("fill from %d to %d at line %d", beginY, endY, x);
	//printf("begin(%f,%f),end(%f,%f),d_outTextureWidth:%d\n", beginUv.x, beginUv.y, endUv.x, endUv.y, d_outTextureWidth);
	//printf("endY:%d,d_outTopTextureHeight:%d,top:%d\n", endY, d_outTopTextureHeight, top);

	for (int y = beginY; y <= top; y++)
	{
		int index = y*d_outTopTextureWidth + x;
		float uvy = beginUv.y + (endUv.y - beginUv.y)*(y - beginY) / (top - beginY);
		d_cudaTopTexture[index] = FillPoint(beginUv.x - 0.5, uvy - 0.5);//tex2D(cudaColorTex, beginUv.x, uvy);
	}
}
__global__ void renderToTexutreTop(int kernelWidth, int kernelHeight)
{
	const int index = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	if (index > kernelWidth)
		return;
	if (index > d_imageWidth)
	{
		FillLine(index);
		return;
	}
	//if (index != 512)
	//	return;
	int listIndex = index;
	int rowLength = d_imageWidth;
	ListNote currentNote = *((ListNote*)&d_cudaPboBuffer[listIndex]);
	int texEnd = 0;
	int texBegin = 0;
	int fillBegin = 0;
	int fillEnd = 0;
	int acuumPixel = 0, span = 0;
	//printf("begin:%d,end%d,index:%d\n", currentNote.beginIndex, currentNote.endIndex, currentNote.nextPt);
	/*while (currentNote.nextPt != 0)
	{
	currentNote = d_listBuffer[currentNote.nextPt];
	rowLength += currentNote.endIndex - currentNote.beginIndex;
	//printf("begin:%d,end%d,index:%d,length:%d\n", currentNote.beginIndex, currentNote.endIndex, currentNote.nextPt,rowLength);
	}*/
	//printf("printf:%d\n", rowLength);
	float factor = d_imageWidth*1.0 / rowLength;
	currentNote = *((ListNote*)&d_cudaPboBuffer[listIndex]);
	int leftEdgeIndex = 0;
	while (currentNote.nextPt != 0)
	{


		currentNote = d_listBuffer[currentNote.nextPt];
		//printf("current:b:%d,e:%d,n:%d,leftEdge:%d\n", currentNote.beginIndex, currentNote.endIndex, currentNote.nextPt, currentNote.leftEdge);

		texEnd = currentNote.endIndex;
		span = currentNote.endIndex - currentNote.beginIndex + 1;
		leftEdgeIndex = currentNote.leftEdge;
		fillBegin = texBegin + acuumPixel;
		fillEnd = texEnd + acuumPixel;
		FillSpanTop(fillBegin*factor, fillEnd*factor, index, toUv(index, texBegin), toUv(index, texEnd));  //for ѭ��������ҿ�
		FillVolumnTop((fillEnd)*factor, (fillEnd + span)*factor, index, texEnd, leftEdgeIndex);

		acuumPixel += span;
		texBegin = currentNote.endIndex;
		//printf("texBegin:%d,acuumPixel:%d,n:%d\n", texBegin, acuumPixel);

	}
	fillBegin = texBegin + acuumPixel;
	//printf("final:(%d,%d) u(%f,%f)\n", fillBegin, d_imageWidth + span, toUv(index, texBegin).y, toUv(index,d_imageWidth).y);
	FillSpanTop(fillBegin*factor, (d_imageWidth + acuumPixel)*factor, index, toUv(index, texBegin), toUv(index, d_imageWidth - 1));

}

// �ڵ�y �е�beginX ��endXֱ�Ӽ�¼�пյ������λ����ϢleftEdge������������߽߱磬endUv���������������ߵ���Ȳ�ֵ
__device__ void FillVolumn(int beginX, int endX, int y, int endUv, int leftEdge, int accumIndex, int  noteIndex)
{
	int top = min(endX, d_outTextureWidth - 1);
	//printf("volumn begin:%d,end:%d,top:%d\n",beginX,endX,top);
	float2 beforeEdgeUv = toUv(endUv - 1, y);
	float3 beforePos = make_float3(tex2D(cudaPosTex, beforeEdgeUv.x, beforeEdgeUv.y));
	float2 endEdgeUv = toUv(endUv, y);
	float3 endPos = make_float3(tex2D(cudaPosTex, endEdgeUv.x, endEdgeUv.y));
	// ��¼�ߵ�
	float2 leftEdgeUv = toUv(leftEdge + 1, y);
	float3 leftEdgePos = make_float3(tex2D(cudaPosTex, leftEdgeUv.x, leftEdgeUv.y));
	float3 eoc_pos = d_eocPos;
	/*printf("endPos:(%f,%f,%f)\n", endPos.x, endPos.y, endPos.z);
	printf("beforePos:(%f,%f,%f)\n", beforePos.x, beforePos.y, beforePos.z);
	printf("leftEdgePos:(%f,%f,%f)\n", leftEdgePos.x, leftEdgePos.y, leftEdgePos.z);
	printf("camera:(%f,%f,%f)\n", d_cameraPos.x, d_cameraPos.y, d_cameraPos.z);
	printf("eoc:(%f,%f,%f)\n", eoc_pos.x, eoc_pos.y, eoc_pos.z);*/

	//for (int i = 0; i < 4; i++)
	//	printf("(%f,%f,%f,%f)\n", d_modelView[4 * i + 0], d_modelView[4 * i + 1], d_modelView[4 * i + 2], d_modelView[4 * i + 3]);
	for (int x = beginX; x <= top; x++)
	{
		int lenght = (top + 1 - beginX);
		float ratio = (x * 1.0f - beginX*1.0f) / lenght;
		float3 realPos = projective_interpo(beforePos, endPos, d_modelViewRight, ratio);
		int index = y*d_outTextureWidth + x;
		float dis = distance(leftEdgePos, realPos, eoc_pos);
		d_cudaTexture[index] = make_float4(dis, realPos.x, realPos.y, realPos.z);
		//printf("x:%d,realpos(%f,%f,%f)\n", x, realPos.x, realPos.y, realPos.z);
		int originMappos = y*d_imageWidth + accumIndex - lenght;
		d_map_buffer[originMappos].y = x;
		d_map_buffer[originMappos].z = noteIndex;
		accumIndex++;
	}
}
// �ڵ�y �е�beginX ��endX������ ����һ���������ԭ��ͼ���beginUV��endUV����һ����ͼ��accumIndexX ��¼ԭ��ͼ����ͼ���ӳ��
__device__ void FillSpan(int beginX, int endX, int y, float2 beginUv, float2 endUv, int* accumIndexX) //beginUv ����toUv����
{
	int top = min(endX, d_outTextureWidth - 1);
	for (int x = beginX; x <= top; x++)
	{
		int index = y*d_outTextureWidth + x;
		float uvx = beginUv.x + (endUv.x - beginUv.x)*(x - beginX) / (top - beginX);
		d_cudaTexture[index] = tex2D(cudaColorTex, uvx, beginUv.y);
		//printf("write uv(%f,%f)\n",uvx,beginUv.y);
		//��¼ӳ���ϵ
		int originMappos = y*d_imageWidth + *accumIndexX;
		//printf("x:%d,mappedPos:%d\n", x, *accumIndexX);
		d_map_buffer[originMappos].x = x;
		*accumIndexX += 1;
	}
}
__global__ void renderToTexutre(int kernelWidth, int kernelHeight)
// ����Ҫ�Ķ�
{
	int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	if (y > kernelHeight)
		return;
	//if (y != 837)
	 //	return;
	int listIndex = y;
	int rowLength = d_imageWidth;
	int texEnd = 0;
	int texBegin = 0;
	int fillBegin = 0;
	int fillEnd = 0;
	int acuumPixel = 0, span = 0;
	//("begin:%d,end%d,index:%d\n", currentNote.beginIndex, currentNote.endIndex, currentNote.nextPt);*	printf("init:%d\n", d_cudaPboBuffer[listIndex].x);
	/*while (currentNote.nextPt != 0)
	{
	currentNote = d_listBuffer[currentNote.nextPt];
	rowLength += currentNote.endIndex - currentNote.beginIndex;
	//printf("begin:%d,end%d,index:%d,length:%d\n", currentNote.beginIndex, currentNote.endIndex, currentNote.nextPt,rowLength);
	}*/
	//printf("printf:%d\n", rowLength);
	int accum_index = 0;  // ��¼�ۼ�
	float factor = d_imageWidth*1.0 / rowLength;
	ListNote currentNote = *((ListNote*)&d_cudaPboBuffer[listIndex]);
	int leftEdgeIndex = 0;
	
	while (currentNote.nextPt != 0)
	{
		/*if (listIndex == 836)
		{
			ListNote currentNote2 = *((ListNote*)&d_cudaPboBuffer[listIndex]);

			printf("Render:next %d end:%d begin:%d,\n", currentNote2.nextPt, currentNote2.endIndex, currentNote2.beginIndex);
		}*/
		int noteIndex = currentNote.nextPt;
		currentNote = d_listBuffer[currentNote.nextPt];
		//	printf("current:b:%d,e:%d,n:%d,leftEdge:%d\n", currentNote.beginIndex, currentNote.endIndex, currentNote.nextPt, currentNote.leftEdge);
		texEnd = currentNote.endIndex;
		span = currentNote.endIndex - currentNote.beginIndex + 1;
		leftEdgeIndex = currentNote.leftEdge;
		fillBegin = texBegin + acuumPixel;
		fillEnd = texEnd + acuumPixel;
		FillSpan(fillBegin*factor, fillEnd*factor, y, toUv(texBegin, y), toUv(texEnd, y), &accum_index);  //�ڲ� for ѭ��������ұ�
		FillVolumn((fillEnd + 1)*factor, (fillEnd + span)*factor, y, texEnd + 1, leftEdgeIndex, accum_index, noteIndex);//�ڲ� for ѭ��������ұ�
		acuumPixel += span;
		texBegin = currentNote.endIndex + 1;
	}
	fillBegin = texBegin + acuumPixel;
	//printf("final:(%d,%d) u(%f,%f)\n", fillBegin, d_imageWidth + span, toUv(texBegin, y).x, toUv(d_imageWidth - 1, y).x);
	FillSpan(fillBegin*factor, (d_imageWidth + acuumPixel)*factor, y, toUv(texBegin, y), toUv(d_imageWidth - 1, y), &accum_index);


}
ListNote *device_data, *device_top_data = NULL;
int atomBuffer = 1; // ԭ�Ӽ�����1��ʼ��0��Ϊ�սڵ��ʶλ
#ifdef DEBUG
ListNote *host_data = NULL;
#endif
extern void cudaInit(int height, int width, int k, int rowLarger)
{
	checkCudaErrors(hipMalloc(&device_data, height*k*sizeof(ListNote)));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_listBuffer), &device_data, sizeof(ListNote*)));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_atomic), &atomBuffer, sizeof(int)));
	checkCudaErrors(hipMemset(device_data, 0, height*k*sizeof(ListNote)));
	//checkCudaErrors(hipMemset(cuda_TexturePbo_buffer, 0, width* height*rowLarger*sizeof(float4)));
#ifdef DEBUG
	checkCudaErrors(hipHostMalloc(&host_data, height*k*sizeof(ListNote)));
#endif
	checkCudaErrors(hipMalloc(&modelView, 16 * sizeof(float)));
	checkCudaErrors(hipMalloc(&proj, 16 * sizeof(float)));

	checkCudaErrors(hipMalloc(&modelViewRight, 16 * sizeof(float)));

	checkCudaErrors(hipMalloc(&cuda_map_buffer, width*height * sizeof(float4)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_map_buffer), &cuda_map_buffer, sizeof(float*)));

	//host_data = (ListNote*)malloc(height*k*sizeof(ListNote));
	//memset(host_data, 0, height*k*sizeof(ListNote));
	//checkCudaErrors(hipMemcpy((void *)device_data, (void *)host_data, height * k * sizeof(ListNote), hipMemcpyDeviceToHost));




}
extern "C" void countRow(int width, int height, Camera * pCamera, Camera * pEocCam, Camera * pEocTopCamera)
{
	checkCudaErrors(hipMemset(cuda_TexturePbo_buffer, 0, ROWLARGER*width*height*sizeof(float4)));
	checkCudaErrors(hipMemset(cuda_top_TexturePbo_buffer, 0, ROWLARGER*width*ROWLARGER*height*sizeof(float4)));
	checkCudaErrors(hipMemset(cuda_map_buffer, 0, width*height*sizeof(float4)));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cameraPos), &pCamera->getCameraPos(), 3 * sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_eocPos), &pEocCam->getCameraPos(), 3 * sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_eocTopPos), &pEocTopCamera->getCameraPos(), 3 * sizeof(float)));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_rightRD), &pEocCam->getDirectionR(), 3 * sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_topRD), &pEocTopCamera->getDirectionR(), 3 * sizeof(float)));


	checkCudaErrors(hipMemcpy(modelView, pCamera->getModelViewMat(), 16 * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_modelView), &modelView, sizeof(float*)));
	checkCudaErrors(hipMemcpy(proj, pCamera->getProjection(), 16 * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_porj), &proj, sizeof(float*)));

	checkCudaErrors(hipMemcpy(modelViewRight, pEocCam->getModelViewMat(), 16 * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_modelViewRight), &modelViewRight, sizeof(float*)));


	hipEvent_t begin_t, end_t;
	checkCudaErrors(hipEventCreate(&begin_t));
	checkCudaErrors(hipEventCreate(&end_t));

	hipEventRecord(begin_t, 0);

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_atomic), &atomBuffer, sizeof(int)));
	checkCudaErrors(hipMemset(cuda_PBO_Buffer, 0, height*sizeof(ListNote)));

	dim3 blockSize(1, 16, 1);
	dim3 gridSize(1, height / blockSize.y, 1);
	countRowKernel << <gridSize, blockSize >> >(1, height);
	hipEventRecord(end_t, 0);
	hipEventSynchronize(end_t);
	float costtime;
	checkCudaErrors(hipEventElapsedTime(&costtime, begin_t, end_t));

	renderToTexutre << <gridSize, blockSize >> >(1, height);
	/**/
	//top Camera
	//checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_atomic), &atomBuffer, sizeof(int)));
	//checkCudaErrors(hipMemset(cuda_PBO_Buffer, 0, height*sizeof(ListNote)));

	dim3 gridSizeT(1, ROWLARGER* width / blockSize.y, 1);
	countRowKernelTop << <gridSizeT, blockSize >> >(width, 1);
	renderToTexutreTop << <gridSizeT, blockSize >> >(ROWLARGER*width, 1);

	checkCudaErrors(hipEventDestroy(begin_t));
	checkCudaErrors(hipEventDestroy(end_t));

#ifdef DEBUG
	/*
	int arraySize = 0;
	checkCudaErrors(hipMemcpy((void *)host_data, (void *)device_data, height*10*sizeof(ListNote), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpyFromSymbol(&arraySize, HIP_SYMBOL(d_atomic), sizeof(int)));

	for (int i = 0; i < arraySize; i++)
	{
	printf("b:%d,n:%d,next:%d\n", host_data[i].beginIndex, host_data[i].endIndex, host_data[i].nextPt);
	}*/
#endif

}

extern "C"  void cudaRelateTex(CudaTexResourse * pResouce)
{

	hipArray *tmpcudaArray;
	hipGraphicsResource ** pCudaTex = pResouce->getResPoint();
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&tmpcudaArray, *pCudaTex, 0, 0));
	int w = pResouce->getWidth();
	int h = pResouce->getHeight();
	if (occluderbuffer_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cudaOccuderTex, tmpcudaArray, channelDesc));
		cudaOccuderTex.filterMode = hipFilterModeLinear;
	}
	else if (edgebuffer_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cudaEdgeTex, tmpcudaArray, channelDesc));
		cudaEdgeTex.filterMode = hipFilterModePoint;
	}
	else if (color_t == pResouce->getType())
	{

		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_imageWidth), &w, sizeof(int)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_imageHeight), &h, sizeof(int)));
		checkCudaErrors(hipBindTextureToArray(cudaColorTex, tmpcudaArray, channelDesc));
		cudaColorTex.filterMode = hipFilterModePoint;
	}
	else if (pos_t == pResouce->getType())
	{
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_imageWidth), &w, sizeof(int)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_imageHeight), &h, sizeof(int)));
		checkCudaErrors(hipBindTextureToArray(cudaPosTex, tmpcudaArray, channelDesc));
		cudaPosTex.filterMode = hipFilterModeLinear;

	}
	else if (occluderTopbuffer_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cudaTopOccuderTex, tmpcudaArray, channelDesc));
		cudaTopOccuderTex.filterMode = hipFilterModePoint;
	}
	else if (optixColorTex_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(optixColorTex, tmpcudaArray, channelDesc));
		optixColorTex.filterMode = hipFilterModePoint;
	}
	else if (posBlend_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(posBlendTex, tmpcudaArray, channelDesc));
		posBlendTex.filterMode = hipFilterModeLinear;
	}
	else if (normal_t == pResouce->getType())
	{

		checkCudaErrors(hipBindTextureToArray(cudaNormalTex, tmpcudaArray, channelDesc));
		cudaNormalTex.filterMode = hipFilterModePoint;
	}
}

extern "C" void cudaRelateArray(CudaPboResource * pResource)
{
	size_t numBytes;
	hipGraphicsResource ** pCudaTex = pResource->getResPoint();
	int w = pResource->getWidth();
	int h = pResource->getHeight();
	if (list_e == pResource->getType())
	{
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_PBO_Buffer, &numBytes, *pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaPboBuffer), &cuda_PBO_Buffer, sizeof(ListNote*)));
	}
	else if (to_optix_t == pResource->getType())
	{
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_outTextureWidth), &w, sizeof(int)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_outTextureHeigh), &h, sizeof(int)));

		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_TexturePbo_buffer, &numBytes, *pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaTexture), &cuda_TexturePbo_buffer, sizeof(float4*)));
	}
	else  if (top_float4_t == pResource->getType())
	{
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_outTopTextureWidth), &w, sizeof(int)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_outTopTextureHeight), &h, sizeof(int)));

		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_top_TexturePbo_buffer, &numBytes, *pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaTopTexture), &cuda_top_TexturePbo_buffer, sizeof(float4*)));

	}
	else if (construct_t == pResource->getType())
	{
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_construct_width), &w, sizeof(int)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_construct_height), &h, sizeof(int)));

		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_construct_texturePbo_buffer, &numBytes, *pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cuda_construct_texture), &cuda_construct_texturePbo_buffer, sizeof(float4*)));
	}
}
void mapConstruct(Camera * pReconstructCamer)
{

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_construct_cam_pos), &pReconstructCamer->getCameraPos(), 3 * sizeof(float)));
	checkCudaErrors(hipMemcpy(modelView_construct, pReconstructCamer->getModelViewMat(), 16 * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_modelView_construct), &modelView_construct, sizeof(float*)));
	checkCudaErrors(hipMemcpy(project_construct, pReconstructCamer->getProjection(), 16 * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_project_construct), &project_construct, sizeof(float*)));
	nv::matrix4f invModelView = inverse(nv::matrix4f(pReconstructCamer->getModelViewMat()));
	checkCudaErrors(hipMemcpy(modelView_inv, invModelView.get_value(), 16 * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_modeView_inv_construct), &modelView_inv, sizeof(float*)));

	nv::vec2f bbmin = nv::vec2f(pReconstructCamer->getImageMin().x, pReconstructCamer->getImageMin().y);
	nv::vec2f bbmax = nv::vec2f(pReconstructCamer->getImageMax().x, pReconstructCamer->getImageMax().y);
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_bbmin), &bbmin, 2 * sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_bbmax), &bbmax, 2 * sizeof(float)));
	nv::vec2f tc = nv::vec2f(412, 512) / nv::vec2f(1024.0, 1024.0);
	nv::vec2f xy = bbmin + (bbmax - bbmin)*tc;
	nv::vec4f temp;
	MutiMatrix((float*)nv::vec4f(xy, -1, 1), (float*)invModelView.get_value(), (float*)& temp);// *;
	//MutiMatrix((float*)&modelViewValue, (float*)invModelView.get_value(), (float*)& temp);// *;

	// nv::vec4f fuck = invModelView * nv::vec4f(modelViewValue);
	//nv::vec4f beginPoint = nv::vec4f(15.402321, -17.913536, -52.650398,1);

	//beginPoint = nv::vec4f(-4.31230021, -17.7475834, -47.2878876, 1);
	//beginPoint = nv::vec4f( nv::vec3f(temp), 1);
	nv::vec4f temp1 = nv::matrix4f(pReconstructCamer->getModelViewMat())* temp;
	nv::vec4f final = nv::matrix4f(pReconstructCamer->getProjection())* temp1;
	final /= final.w;
	final.x = final.x*0.5 + 0.5;
	final.y = final.y*0.5 + 0.5;
	final.x *= 1024;
	final.y *= 1024;


}

__device__ float3 getImagePos(float2 tc, float* modelViewInv)
{
	float2 xy = d_bbmin + (d_bbmax - d_bbmin)*tc;
	xy = xy;
	float4 temp = MutiMatrixN(modelViewInv, make_float4(xy.x, xy.y, -1, 1));// must
	temp = temp / temp.w;
	return make_float3(temp.x, temp.y, temp.z);
}
__device__ float3 toFloat3(float4 inValue)
{
	return make_float3(inValue.x / inValue.w, inValue.y / inValue.w, inValue.z / inValue.w);
}

__device__ int intersectCameraID(float3 posW, float3 directionW, float3 cameraPos, ListNote currentNote, int yIndex, bool isRayUp, int occludedObjId,
								float* modelView,// ��ѯmodelView ����µ����
								float4& intersectColor, float2& exitTC, float3& exitWorldPos)
{
	int texEnd = currentNote.endIndex;  // ������ұ߽߱�-��ֵ
	int texBegin = currentNote.beginIndex;
	int span = texEnd + 1 - texBegin;
	int currentObjectId = nearestInt(tex2D(cudaNormalTex, texEnd - span / 2.0, yIndex + 0.5).w);
	//printf("current obj fetch (%f,%f)\n", texEnd + span / 2.0, yIndex + 0.5);
	//printf("note mid obj Id:%d\n", currentObjectId);
	//printf("texBegin,span(%d,%d)\n", texBegin, span);
	if (currentObjectId != occludedObjId)
	{
		//printf("objectId not same\n");
		return OHTEROBJECT;
	}
#define GAP 0.01
	float exitY, enterY;
	if (isRayUp)
	{
		exitY = yIndex + 1.0 - GAP;
		enterY = yIndex + GAP;
	}
	else
	{
		enterY = yIndex + 1.0 - GAP;
		exitY = yIndex + GAP;
	}
	float2 beforeEnterTc = make_float2(texEnd + 0.5, enterY);                 //left
	float3 beforeEntorPos = make_float3(tex2D(cudaPosTex, beforeEnterTc.x, beforeEnterTc.y));
	float2 endEntorTc = make_float2(texEnd + 1 + 0.5, enterY);
	float3 endEntorPos = make_float3(tex2D(cudaPosTex, endEntorTc.x, endEntorTc.y));
	float2 beforeExitTc = make_float2(texEnd + 0.5, exitY);
	float3 beforeExitPos = make_float3(tex2D(cudaPosTex, beforeExitTc.x, beforeExitTc.y));
	float2 endExitTc = make_float2(texEnd + 1 + 0.5, exitY);
	float3 endExitPos = make_float3(tex2D(cudaPosTex, endExitTc.x, endExitTc.y));
	float enter_projRatio, exit_projRatiok;
	float3 enterReservedPos, exitReservedPos, _;
	bool f_;
	rayIntersertectTriangle(posW, normalize(directionW), cameraPos, beforeEntorPos, endEntorPos, d_modelViewRight, span, &enterReservedPos, &_, f_, enter_projRatio, _);
	rayIntersertectTriangle(posW, normalize(directionW), cameraPos, beforeExitPos, endExitPos, d_modelViewRight, span, &exitReservedPos, &_, f_, exit_projRatiok, _);
	
	float2 camera1Entertc = getCameraTc(enterReservedPos, d_modelViewRight, d_porj);
	//printf("camera1 entertc:(%f,%f)\n", 1024 * camera1Entertc.x, 1024 * camera1Entertc.y);
	float2 camera1EXittc = getCameraTc(exitReservedPos, d_modelViewRight, d_porj);
	//printf("camera1 exittc:(%f,%f)\n", 1024 * camera1EXittc.x, 1024 * camera1EXittc.y);
	
	float4 temp = MutiMatrixN(modelView, make_float4(enterReservedPos, 1));
	float enterZ = -temp.z;
	temp = MutiMatrixN(modelView, make_float4(exitReservedPos, 1));
	float exitZ = -temp.z;
	float step = (enter_projRatio > exit_projRatiok) ? -1.0 : 1.0;
	float enterP = min(1, max(0, enter_projRatio));
	float exitP = min(1, max(0, exit_projRatiok));
	//printf("enterP,enterP(%f,%f)\n", enterP, exitP);                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                    
	float dpdx = (repo(exitZ) - repo(enterZ)) / (exit_projRatiok - enter_projRatio);
	bool isInLoop = false;
	float lastSearch, currentZ;
	float4 color;
	for (float tex = texBegin + span* enterP; tex < texBegin + span* exitP; tex += step)
	{
		float ratioOneD = (tex - texBegin) / span;
		currentZ = repo(repo(enterZ) + (ratioOneD - enter_projRatio)*dpdx);
		//printf("ratioOneD:%f,test tx:%f,currentZ:%f\n", ratioOneD, tex, currentZ);
		if (noMappedPosition(make_float2(tex, yIndex + 0.5), &color))
		{
			float zOnTex = color.w;
			//printf("mapped Z:%f\n", zOnTex);
			if (currentZ > zOnTex)
			{
				//printf("intersect\n");
				intersectColor = color;
				return INTERSECT;

			}
		}
		else
		{
			exitWorldPos = exitReservedPos;
			return OUTOCCLUDED;
		}
		lastSearch = tex;
		isInLoop = true;
	}
	/*
	exitTC.x = (texBegin + span* exitP) / d_imageWidth;
	exitWorldPos = exitReservedPos;
	if (isInLoop)
	{
		//printf("last x:%f\n", lastSearch);
		exitTC.x = lastSearch / d_imageWidth;
		float zRatio = (currentZ - enterZ) / (exitZ - enterZ);
		exitWorldPos = lerp(enterReservedPos, exitReservedPos, zRatio);
		//printf("zRatio:%f\n", zRatio);
		float2 outTc = getCameraTc(exitWorldPos, d_modelView, d_porj);
		//printf("outTc :(%f,%f)\n", 1024 * outTc.x, 1024 * outTc.y);
	}
	float2 outtc = getCameraTc(exitWorldPos, d_modelView, d_porj);
	*/
	//printf(" missingNote\n");
	return MISSINGNOTE;



#undef GAP
	
}
// �����objectId��������ڵ���������󽻽��
__device__ int isIntersectLineWithObjectId(float3 posW, float3 directionW, float3 cameraPos, int lineNum, bool isRayUp, int occudedObjId,
							float* modelView,float3& outPos,float4& resultColor)
{
	ListNote currentNote = *((ListNote*)&d_cudaPboBuffer[lineNum]);

	//printf("Render:next %d end:%d begin:%d,\n", currentNote.nextPt, currentNote.endIndex, currentNote.beginIndex);
	int leftEdgeIndex = 0;
	bool hasObjectNote = false;
	while (currentNote.nextPt != 0)
	{
		int noteIndex = currentNote.nextPt;
		currentNote = d_listBuffer[currentNote.nextPt];


		//printf("intersect with a line\n");
		float2 _;
		int result = intersectCameraID(posW, directionW, cameraPos, currentNote, lineNum, isRayUp, occudedObjId, modelView, resultColor, _, outPos);
		
		if (result != OHTEROBJECT)
		{
			//����ҵ�����ͬobj
			//printf("obj found\n");
			hasObjectNote = true;
		}
		if (OUTOCCLUDED == result)
		{
			return OUTOCCLUDED;
		}
		else if (INTERSECT == result)
		{
			return INTERSECT;
		}
		else if (currentNote.nextPt == 0)
		{
			//���һ���ڵ�
			break;
		}
		else if (OHTEROBJECT == result || MISSINGNOTE == result)
		{
			//test other note in the same line
			currentNote = *((ListNote*)&d_cudaPboBuffer[currentNote.nextPt]);
		}
	}
	
	if (hasObjectNote)
	{
		//����û���յ�
		return MISSINGNOTE;
	}
	return NOOBJECTNOTE;
}
//#define RAYISUP 0
//#define RAYOUT 1
//#define RAYISUNDER 2
//���궼��0-1�ռ�
__device__ int rayBelowMainTex(float n, int stepN,float2 projStart,float2 interval,float rayStartz, float rayEndZ,float2 &tc)
{
	float alpha = n / stepN;
	tc = projStart + interval* n / stepN;
	float currRayPointZ = 1 / ((1 - alpha)*(1 / rayStartz) + (alpha)*(1 / rayEndZ));
	float currSamplePointZ = colorTextreNorTc(tc).w;
	if (tc.x>1 || tc.x<0 || tc.y<0 || tc.y>1 || currSamplePointZ > 0)
	{
		return 0;
	}
	// ��Ϊ��-ֵ�����ߵı�ͼƬ���Զ������ζ��zҪС
	else if (currRayPointZ <= currSamplePointZ)
		return RAYISUNDER;
	else
		return RAYISUP;
}
__device__ int intersectTexRay(float3 posW, float3 directionW, float4& oc)
{
	float2 d_mapScale = 1.0 / make_float2(d_construct_width, d_construct_height);
	float3 rayStart, rayEnd;
	float4 color;
	//printf("posW:(%f,%f,%f,1)\n", posW.x, posW.y, posW.z);
	float4 posWE = MutiMatrixN(d_modelView, make_float4(posW, 1));
	float4 temp2 = MutiMatrixN(d_porj, posWE);
	temp2 = temp2 / temp2.w;
	//printf("temp2:%f,%f", (temp2.x*0.5 + 0.5) * 1024, (temp2.y*0.5 + 0.5) * 1024);
	float3 posW3 = toFloat3(posWE);
	float4 temp = MutiMatrixN(d_modelView, make_float4(directionW, 0));
	float3 RE = normalize(make_float3(temp.x, temp.y, temp.z));
	//printf("RE:(%f,%f,%f,1)\n", RE.x, RE.y, RE.z);
	float epison = 10.2;
	rayStart = posW3 + RE*epison;

	float max_rfl = 370;//far*diffuseColor.w;
	rayEnd = posW3 + RE*max_rfl;

	//p.color0.xy = tc;
	if (rayEnd.z>0)
	{
		float step = -posW3.z / RE.z;
		rayEnd = posW3 + RE*(step - 1);
	}
	temp = MutiMatrixN(d_porj, make_float4(rayStart, 1));

	float3 projStart = toFloat3(temp);
	temp = MutiMatrixN(d_porj, make_float4(rayEnd, 1));
	float3 projEnd = toFloat3(temp);

	projStart.x = 0.5*projStart.x + 0.5;
	projEnd.x = 0.5*projEnd.x + 0.5;
	projStart.y = 0.5*projStart.y + 0.5;
	projEnd.y = 0.5*projEnd.y + 0.5;
	

	//printf("projStart(%f,%f),projEnd(%f,%f)\n", (projStart.x) * 1024, (projStart.y) * 1024, projEnd.x * 1024, projEnd.y * 1024);
	
	
	//oc = make_float4(projStart.x,projStart.y,projStart.z,0.7);	
	//return 1;
	float2 interval = make_float2(projEnd.x, projEnd.y) - make_float2(projStart.x, projStart.y);
	int stepN;
	//printf("interval:(%f,%f)\n", interval.x, interval.y);
	//printf("1024*d_mapScale:(%f,%f)\n", d_mapScale.x*1024,d_mapScale.y*1024);
	if (abs(interval.x)>abs(interval.y))
		stepN = abs(interval.x) / d_mapScale.x + 1;
	else
		stepN = abs(interval.y) / d_mapScale.y + 1;

	float currSamplePointZ, currRayPointZ, prevSamplePointZ, prevRayPointZ;
	float3 currSamplePoint, currRayPoint;
	float n = 0;
	float2 tc;
	bool isNotValid = true;

	
	//printf("interval*1024(%f,%f)\n", (interval.x) * 1024, (interval.y) * 1024);
	//printf("stepN:%d\n", stepN);
	int prevState,currentRayState = rayBelowMainTex(n, stepN, make_float2(projStart.x, projStart.y), interval, rayStart.z, rayEnd.z, tc);
	if (RAYOUT == currentRayState)
	{
		return false;// û������ռ���
	}
	if (stepN<2)
	{
		oc = colorTextreNorTc(make_float2(projStart.x, projStart.y) + interval / 2);
		return 1;
	}
	for (; n <= stepN;	n += 1)
	{
		prevState = currentRayState;
		currentRayState = rayBelowMainTex(n, stepN, make_float2(projStart.x, projStart.y), interval, rayStart.z, rayEnd.z, tc);
		if (RAYOUT == currentRayState)
		{
			return false;// û������ռ���
		}
		if (RAYISUNDER == currentRayState && RAYISUP == prevState)
		{
			color = colorTextreNorTc(tc);
			float lastAlpha = 0;
			if (n >= 1)
				lastAlpha = (float)(n - 1) / stepN;
			float2 lastTc = make_float2(projStart.x, projStart.y) + interval* lastAlpha;
			int previewsN = n;
			if (isTracingEdge(lastTc))// ��������ڶ������ڱ����ϣ�Ҳ���ǽ����ڵ���
			{
				int stepY = abs(interval.y) / d_mapScale.y + 1;
				float3 exitPos;
				bool rayAdvanced = false;
				if (isOccluedeArea(tc))//search in occluded area
				{
					//�ҵ��ڵ����bojectId
					float2 nonNorTc = tc* make_float2(d_imageWidth, d_imageHeight);
					int objectId = nearestInt(tex2D(cudaNormalTex, nonNorTc.x, nonNorTc.y).w);
					bool isUp = interval.y > 0;
					int startLineNum = floor(tc.y*d_imageHeight);  // �����6.6 �У�����6������
					float3 outPos;
					int result = isIntersectLineWithObjectId(posW, directionW, d_eocPos, startLineNum, isUp, objectId, d_modelViewRight, outPos, oc);
					int lineId = startLineNum + (isUp ? 1 : -1);// lineId �洢����Ҫ��������һ��
					n = previewsN;
					n += (float)stepN / stepY;
					while (result == MISSINGNOTE)// �����һ����û���󽻵�MISS,����û�з���
					{
						
						result = isIntersectLineWithObjectId(posW, directionW, d_eocPos, lineId, isUp, objectId, d_modelViewRight, outPos, oc);
						lineId = lineId + (isUp ? 1 : -1);
						n += (float)stepN / stepY;	

					}
					if (result == INTERSECT)
					{
						return 1;
					}
					else if (OUTOCCLUDED == result)//�������һ���»ص������
					{
						//ͨ��lineId �ĺ������ж�n����n�ĵط���Ϊ���е�һ��ƫ�ƴ���
						float newN;
#define GAP 0.01
						newN = stepN * abs(lineId + GAP - projStart.y / d_mapScale.y) / (abs(interval.y) / d_mapScale.y);
						n = max(newN, previewsN);
						currentRayState = rayBelowMainTex(n, stepN, make_float2(projStart.x, projStart.y), interval, rayStart.z, rayEnd.z, tc);
#undef GAP
						continue;// ��һ��Ѱ��
					}
					else
					{
						continue;
					}
					
				}
				else
				{
					//printf("no intersection in occluded area\n");
					return 0;
				}
			}
			//�����ӽ������ҵ��˽���
			color.w = 1;
			oc = color;
			//printf("main camera intersection found\n");
			return 1;
		}
	}
	// ��������Χ���Ҳ���
	return 0;

}

__global__ void construct_kernel(int kernelWidth, int kernelHeight)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= kernelWidth || y >= kernelHeight)
		return;
	//if (x != 420 || y != 840)
	//    return;
	//if ( y >= 470||x>=414)
	//	return;
	//printf("test:x%d,y:%d\n", x, y);
	const int index = y*kernelWidth + x;
	float2 tc = make_float2(x + 0.5, y + 0.5) / make_float2(kernelWidth, kernelHeight);
	float3 beginPoint = getImagePos(tc, d_modeView_inv_construct);
	//printf("beginPoint:(%f,%f,%f)\n", beginPoint.x, beginPoint.y, beginPoint.z);
	float3 viewDirection = beginPoint - d_construct_cam_pos;

	//printf("viewDirection:(%f,%f,%f)\n", viewDirection.x, viewDirection.y, viewDirection.z);
	float4 outColor;
	if (intersectTexRay(beginPoint, viewDirection, outColor))
	{
		//printf("here outcolor(%f,%f,%f,%f)\n", outColor.x, outColor.y, outColor.z, outColor.w);
		d_cuda_construct_texture[index] = make_float4(outColor.x, outColor.y, outColor.z, 1);//tex2D(cudaColorTex, x, y);
	}
	else
	{
		d_cuda_construct_texture[index] = make_float4(0, 0, 0, 1);//tex2D(cudaColorTex, x, y);
	}
}
void construct_cudaInit()
{
	checkCudaErrors(hipMalloc(&modelView_construct, 16 * sizeof(float)));
	checkCudaErrors(hipMalloc(&project_construct, 16 * sizeof(float)));
	checkCudaErrors(hipMalloc(&modelView_inv, 16 * sizeof(float)));


}
void cuda_Construct(int width, int height)
{
	dim3 blockSize(16, 16, 1);
	dim3 gridSize(width / blockSize.x, height / blockSize.y, 1);
	construct_kernel << <gridSize, blockSize >> >(width, height);
}
